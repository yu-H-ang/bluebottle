#include "hip/hip_runtime.h"
#include "cuda_Eulerian.h"
#include "cuda_bluebottle.h"
#include "bluebottle.h"
#include "Eulerian.h"
#include "entrySearch.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C"
void cuda_Eulerian_push(void)
{
	int i, j, k;          // iterators
	int ii, jj, kk;       // helper iterators
	int C, CC;            // cell references
	
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		// set up host working arrays for subdomain copy from host to device
		real *nn = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		//real *uu_p = (real*) malloc(dom[dev].Gfx.s3b * sizeof(real));
		//real *vv_p = (real*) malloc(dom[dev].Gfy.s3b * sizeof(real));
		//real *ww_p = (real*) malloc(dom[dev].Gfz.s3b * sizeof(real));
		real *bubv = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		real *cc = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		
		// number density
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					nn[CC] = numden[C];
				}
			}
		}
		/*
		// u_p
		for(k = dom[dev].Gfx.ksb; k < dom[dev].Gfx.keb; k++) {
			for(j = dom[dev].Gfx.jsb; j < dom[dev].Gfx.jeb; j++) {
				for(i = dom[dev].Gfx.isb; i < dom[dev].Gfx.ieb; i++) {
					ii = i - dom[dev].Gfx.isb;
					jj = j - dom[dev].Gfx.jsb;
					kk = k - dom[dev].Gfx.ksb;
					C = i + j * Dom.Gfx.s1b + k * Dom.Gfx.s2b;
					CC = ii + jj * dom[dev].Gfx.s1b + kk * dom[dev].Gfx.s2b;
					uu_p[CC] = u_p[C];
				}
			}
		}
		// v_p
		for(k = dom[dev].Gfy.ksb; k < dom[dev].Gfy.keb; k++) {
			for(j = dom[dev].Gfy.jsb; j < dom[dev].Gfy.jeb; j++) {
				for(i = dom[dev].Gfy.isb; i < dom[dev].Gfy.ieb; i++) {
					ii = i - dom[dev].Gfy.isb;
					jj = j - dom[dev].Gfy.jsb;
					kk = k - dom[dev].Gfy.ksb;
					C = i + j * Dom.Gfy.s1b + k * Dom.Gfy.s2b;
					CC = ii + jj * dom[dev].Gfy.s1b + kk * dom[dev].Gfy.s2b;
					vv_p[CC] = v_p[C];
				}
			}
		}
		// w_p
		for(k = dom[dev].Gfz.ksb; k < dom[dev].Gfz.keb; k++) {
			for(j = dom[dev].Gfz.jsb; j < dom[dev].Gfz.jeb; j++) {
				for(i = dom[dev].Gfz.isb; i < dom[dev].Gfz.ieb; i++) {
					ii = i - dom[dev].Gfz.isb;
					jj = j - dom[dev].Gfz.jsb;
					kk = k - dom[dev].Gfz.ksb;
					C = i + j * Dom.Gfz.s1b + k * Dom.Gfz.s2b;
					CC = ii + jj * dom[dev].Gfz.s1b + kk * dom[dev].Gfz.s2b;
					ww_p[CC] = w_p[C];
				}
			}
		}
		*/
		// bubble volume
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					bubv[CC] = bubvol[C];
				}
			}
		}
		// concentration
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					cc[CC] = concen[C];
				}
			}
		}
		
		// copy from host to device
		checkCudaErrors(hipMemcpy(_numden[dev], nn, sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyHostToDevice));
		//checkCudaErrors(hipMemcpy(_u_p[dev], uu_p, sizeof(real) * dom[dev].Gfx.s3b, hipMemcpyHostToDevice));
		//checkCudaErrors(hipMemcpy(_v_p[dev], vv_p, sizeof(real) * dom[dev].Gfy.s3b, hipMemcpyHostToDevice));
		//checkCudaErrors(hipMemcpy(_w_p[dev], ww_p, sizeof(real) * dom[dev].Gfz.s3b, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(_bubvol[dev], bubv, sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(_concen[dev], cc, sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyHostToDevice));
		
		// free host subdomain working arrays
		free(nn);
		//free(uu_p);
		//free(vv_p);
		//free(ww_p);
		free(bubv);
		free(cc);
	}
}

void cuda_Eulerian_pull(void)
{
	// copy device data to host
	#pragma omp parallel num_threads(nsubdom)
	{
		int i, j, k;          // iterators
		int ii, jj, kk;       // helper iterators
		int C, CC;            // cell references
		
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));

		// host working arrays for subdomain copy from device to host
		real *nn = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		//real *uu_p = (real*) malloc(dom[dev].Gfx.s3b * sizeof(real));
		//real *vv_p = (real*) malloc(dom[dev].Gfy.s3b * sizeof(real));
		//real *ww_p = (real*) malloc(dom[dev].Gfz.s3b * sizeof(real));
		real *bubv = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		real *cc = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		
		// copy from device to host
		checkCudaErrors(hipMemcpy(nn, _numden[dev], sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyDeviceToHost));
		//checkCudaErrors(hipMemcpy(uu_p, _u_p[dev], sizeof(real) * dom[dev].Gfx.s3b, hipMemcpyDeviceToHost));
		//checkCudaErrors(hipMemcpy(vv_p, _v_p[dev], sizeof(real) * dom[dev].Gfy.s3b, hipMemcpyDeviceToHost));
		//checkCudaErrors(hipMemcpy(ww_p, _w_p[dev], sizeof(real) * dom[dev].Gfz.s3b, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(bubv, _bubvol[dev], sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(cc, _concen[dev], sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyDeviceToHost));
		
		// fill in apropriate subdomain (copy back ghost cells)
		// numden
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					numden[C] = nn[CC];
				}
			}
		}
		/*
		// u
		for(k = dom[dev].Gfx.ksb; k < dom[dev].Gfx.keb; k++) {
			for(j = dom[dev].Gfx.jsb; j < dom[dev].Gfx.jeb; j++) {
				for(i = dom[dev].Gfx.isb; i < dom[dev].Gfx.ieb; i++) {
					ii = i - dom[dev].Gfx.isb;
					jj = j - dom[dev].Gfx.jsb;
					kk = k - dom[dev].Gfx.ksb;
					C = i + j * Dom.Gfx.s1b + k * Dom.Gfx.s2b;
					CC = ii + jj * dom[dev].Gfx.s1b + kk * dom[dev].Gfx.s2b;
					u_p[C] = uu_p[CC];
				}
			}
		}
		// v
		for(k = dom[dev].Gfy.ksb; k < dom[dev].Gfy.keb; k++) {
			for(j = dom[dev].Gfy.jsb; j < dom[dev].Gfy.jeb; j++) {
				for(i = dom[dev].Gfy.isb; i < dom[dev].Gfy.ieb; i++) {
					ii = i - dom[dev].Gfy.isb;
					jj = j - dom[dev].Gfy.jsb;
					kk = k - dom[dev].Gfy.ksb;
					C = i + j * Dom.Gfy.s1b + k * Dom.Gfy.s2b;
					CC = ii + jj * dom[dev].Gfy.s1b + kk * dom[dev].Gfy.s2b;
					v_p[C] = vv_p[CC];
				}
			}
		}
		// w
		for(k = dom[dev].Gfz.ksb; k < dom[dev].Gfz.keb; k++) {
			for(j = dom[dev].Gfz.jsb; j < dom[dev].Gfz.jeb; j++) {
				for(i = dom[dev].Gfz.isb; i < dom[dev].Gfz.ieb; i++) {
					ii = i - dom[dev].Gfz.isb;
					jj = j - dom[dev].Gfz.jsb;
					kk = k - dom[dev].Gfz.ksb;
					C = i + j * Dom.Gfz.s1b + k * Dom.Gfz.s2b;
					CC = ii + jj * dom[dev].Gfz.s1b + kk * dom[dev].Gfz.s2b;
					w_p[C] = ww_p[CC];
				}
			}
		}
		*/
		// bubvol
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					bubvol[C] = bubv[CC];
				}
			}
		}
		// concen
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					concen[C] = cc[CC];
				}
			}
		}
		
		// free host subdomain working arrays
		free(nn);
		//free(uu_p);
		//free(vv_p);
		//free(ww_p);
		free(bubv);
		free(cc);
	}
}

extern "C"
void cuda_Eulerian_free(void)
{
	// free device memory on device
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		checkCudaErrors(hipFree(_numden[dev]));
		checkCudaErrors(hipFree(_nextnumden[dev]));
		//checkCudaErrors(hipFree(_u_p[dev]));
		//checkCudaErrors(hipFree(_v_p[dev]));
		checkCudaErrors(hipFree(_w_p[dev]));
		//checkCudaErrors(hipFree(_f_x_coupling_numden[dev]));
		//checkCudaErrors(hipFree(_f_y_coupling_numden[dev]));
		checkCudaErrors(hipFree(_f_z_coupling_numden[dev]));
		
		checkCudaErrors(hipFree(_bubvol[dev]));
		checkCudaErrors(hipFree(_nextbubvol[dev]));
		checkCudaErrors(hipFree(_bubdia[dev]));
		checkCudaErrors(hipFree(_bubdiafz[dev]));
		
		checkCudaErrors(hipFree(_concen[dev]));
		checkCudaErrors(hipFree(_nextconcen[dev]));
		checkCudaErrors(hipFree(_velmag[dev]));
		checkCudaErrors(hipFree(_mdot[dev]));
	}
	
	// free device memory on host
	free(_numden);
	free(_nextnumden);
	//free(_u_p);
	//free(_v_p);
	free(_w_p);
	//free(_f_x_coupling_numden);
	//free(_f_y_coupling_numden);
	free(_f_z_coupling_numden);
	
	free(_bubvol);
	free(_nextbubvol);
	free(_bubdia);
	free(_bubdiafz);
	
	free(_concen);
	free(_nextconcen);
	free(_velmag);
	free(_mdot);
}

extern "C"
void cuda_numberdensity_BC(void)
{
	// CPU threading for multi-GPU
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int threads_z = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		// check whether each subdomain boundary (E, W, N, S, T, B) is
		// an external boundary
		// *********************************************************************
		if(dom[dev].W == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks_n(threads_y, threads_z);
			dim3 numBlocks_n(blocks_y, blocks_z);
			
			// apply BC to numberdensity field for this face
			switch(numdenBC.nW) {
				case PERIODIC:
					BC_p_W_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_W_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].E == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks_n(threads_y, threads_z);
			dim3 numBlocks_n(blocks_y, blocks_z);
			// apply BC to numberdensity field for this face
			switch(numdenBC.nE) {
				case PERIODIC:
					BC_p_E_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_E_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].S == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks_n(threads_z, threads_x);
			dim3 numBlocks_n(blocks_z, blocks_x);
			// apply BC to numberdensity field for this face
			switch(numdenBC.nS) {
				case PERIODIC:
					BC_p_S_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_S_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].N == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks_n(threads_z, threads_x);
			dim3 numBlocks_n(blocks_z, blocks_x);

			// apply BC to numberdensity field for this face
			switch(numdenBC.nN) {
					case PERIODIC:
					BC_p_N_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
				break;
					case NEUMANN:
					BC_p_N_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
				break;
			}
		}
		// *********************************************************************
		if(dom[dev].B == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks_n(threads_x, threads_y);
			dim3 numBlocks_n(blocks_x, blocks_y);
			// apply BC to numberdensity field for this face
			switch(numdenBC.nB) {
				case PERIODIC:
					BC_p_B_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_B_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_B_D<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev], numdenBC.nBD);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].T == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks_n(threads_x, threads_y);
			dim3 numBlocks_n(blocks_x, blocks_y);
			// apply BC to numberdensity field for this face
			switch(numdenBC.nT) {
				case PERIODIC:
					BC_p_T_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_T_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_T_D<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev], numdenBC.nTD);
					break;
			}
		}
		// *********************************************************************
	}
}

extern "C"
void cuda_numberdensity_march(void)
{
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_y = 0;
		int threads_z = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		if(dom[dev].Gcc._jn < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc._jn;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc._kn < MAX_THREADS_DIM)
			threads_z = dom[dev].Gcc._kn;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gcc._jn / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gcc._kn / (real) threads_z);
		
		dim3 dimBlocks_n(threads_y, threads_z);
		dim3 numBlocks_n(blocks_y, blocks_z);
		
		kernel_march_numberdensity<<<numBlocks_n, dimBlocks_n>>>(dt,
		                                                         _dom[dev],
		                                                         _numden[dev],
		                                                         _nextnumden[dev],
		                                                         _u[dev],
		                                                         _v[dev],
		                                                         _w_p[dev]);
		
		kernel_inner_scalarfield_update_x<<<numBlocks_n, dimBlocks_n>>>(_dom[dev],
		                                                                _numden[dev],
		                                                                _nextnumden[dev]);
	}
}


extern "C"
void cuda_compute_particle_velz(void)
{
	// parallelize over CPU threads
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		
		// z-component
		if(dom[dev].Gfz._inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfz._inb;
		else
			threads_x = MAX_THREADS_DIM;
		if(dom[dev].Gfz._jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfz._jnb;
		else
			threads_y = MAX_THREADS_DIM;
		
		blocks_x = (int)ceil((real) dom[dev].Gfz._inb / (real) threads_x);
		blocks_y = (int)ceil((real) dom[dev].Gfz._jnb / (real) threads_y);
		dim3 dimBlocks_z(threads_x, threads_y);
		dim3 numBlocks_z(blocks_x, blocks_y);
		
		// since bubble diameter is a field, here combine all the values in terminal velocity except bubble diameter
		real cons = -1.0/18.0*(rho_f - bubble_density)/mu*g.zm;
		
		kernel_numberdensity_particle_velz<<<numBlocks_z, dimBlocks_z>>>(cons, _w_p[dev], _w[dev], _bubdia[dev], _dom[dev]);
	}
}

extern "C"
void cuda_Eulerian_malloc(void)
{
	// allocate device memory on host
	_numden = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_nextnumden = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	//_u_p = (real**) malloc(nsubdom * sizeof(real*));
	//cpumem += nsubdom * sizeof(real*);
	//_v_p = (real**) malloc(nsubdom * sizeof(real*));
	//cpumem += nsubdom * sizeof(real*);
	_w_p = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	//_f_x_coupling_numden = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	//_f_y_coupling_numden = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_f_z_coupling_numden = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	
	_bubvol = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_nextbubvol = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_bubdia = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_bubdiafz = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	
	_concen = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_nextconcen = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_velmag = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_mdot = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	
	// allocate device memory on device
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		// number density equation
		checkCudaErrors(hipMalloc((void**) &(_numden[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		// _numden_next is to store _numden in next time step, only exist on deivce.
		checkCudaErrors(hipMalloc((void**) &(_nextnumden[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		//checkCudaErrors(hipMalloc((void**) &(_u_p[dev]), sizeof(real) * dom[dev].Gfx.s3b));
		//gpumem += dom[dev].Gfx.s3b * sizeof(real);
		//checkCudaErrors(hipMalloc((void**) &(_v_p[dev]), sizeof(real) * dom[dev].Gfy.s3b));
		//gpumem += dom[dev].Gfy.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_w_p[dev]), sizeof(real) * dom[dev].Gfz.s3b));
		gpumem += dom[dev].Gfz.s3b * sizeof(real);
		//checkCudaErrors(hipMalloc((void**) &(_f_x_coupling_numden[dev]), sizeof(real) * dom[dev].Gfx.s3b));
		//gpumem += dom[dev].Gfx.s3b * sizeof(real);
		//checkCudaErrors(hipMalloc((void**) &(_f_y_coupling_numden[dev]), sizeof(real) * dom[dev].Gfy.s3b));
		//gpumem += dom[dev].Gfy.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_f_z_coupling_numden[dev]), sizeof(real) * dom[dev].Gfz.s3b));
		gpumem += dom[dev].Gfz.s3b * sizeof(real);
		
		// bubble volume
		checkCudaErrors(hipMalloc((void**) &(_bubvol[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_nextbubvol[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_bubdia[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_bubdiafz[dev]), sizeof(real) * dom[dev].Gfz.s3b));
		gpumem += dom[dev].Gfz.s3b * sizeof(real);
		
		// concentration
		checkCudaErrors(hipMalloc((void**) &(_concen[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		// _nextconcen is to store _concen in next time step, only exist on deivce.
		checkCudaErrors(hipMalloc((void**) &(_nextconcen[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		// _velmag is to store the velocity magnitude of the fluid flow,
		// which is used in the source term of mass transfer, only exist on device.
		checkCudaErrors(hipMalloc((void**) &(_velmag[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_mdot[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
	}
}

extern "C"
void cuda_concentration_BC(void)
{
	// CPU threading for multi-GPU
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int threads_z = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		// check whether each subdomain boundary (E, W, N, S, T, B) is
		// an external boundary
		// *********************************************************************
		if(dom[dev].W == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks_n(threads_y, threads_z);
			dim3 numBlocks_n(blocks_y, blocks_z);
			
			// apply BC to concentration field for this face
			switch(concenBC.nW) {
				case PERIODIC:
					BC_p_W_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_W_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].E == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks_n(threads_y, threads_z);
			dim3 numBlocks_n(blocks_y, blocks_z);
			// apply BC to concentration field for this face
			switch(concenBC.nE) {
				case PERIODIC:
					BC_p_E_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_E_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].S == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks_n(threads_z, threads_x);
			dim3 numBlocks_n(blocks_z, blocks_x);
			// apply BC to concentration field for this face
			switch(concenBC.nS) {
				case PERIODIC:
					BC_p_S_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_S_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].N == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks_n(threads_z, threads_x);
			dim3 numBlocks_n(blocks_z, blocks_x);

			// apply BC to concentration field for this face
			switch(concenBC.nN) {
					case PERIODIC:
					BC_p_N_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
				break;
					case NEUMANN:
					BC_p_N_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
				break;
			}
		}
		// *********************************************************************
		if(dom[dev].B == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks_n(threads_x, threads_y);
			dim3 numBlocks_n(blocks_x, blocks_y);
			// apply BC to concentration field for this face
			switch(concenBC.nB) {
				case PERIODIC:
					BC_p_B_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_B_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_B_D<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev], concenBC.nBD);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].T == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks_n(threads_x, threads_y);
			dim3 numBlocks_n(blocks_x, blocks_y);
			// apply BC to concentration field for this face
			switch(concenBC.nT) {
				case PERIODIC:
					BC_p_T_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_T_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_T_D<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev], concenBC.nTD);
					break;
			}
		}
		// *********************************************************************
	}
}

extern "C"
void cuda_compute_coupling_forcing(void)
{
	// parallelize over CPU threads
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));

		int threads_x = 0;
		int threads_y = 0;
		int threads_z = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		int blocks_z = 0;

		// x-component
		if(dom[dev].Gfx._jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfx._jnb;
		else
			threads_y = MAX_THREADS_DIM;

		if(dom[dev].Gfx._knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gfx._knb;
		else
			threads_z = MAX_THREADS_DIM;

		blocks_y = (int)ceil((real) dom[dev].Gfx._jnb / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gfx._knb / (real) threads_z);

		dim3 dimBlocks_x(threads_y, threads_z);
		dim3 numBlocks_x(blocks_y, blocks_z);

		// y-component
		if(dom[dev].Gfy._knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gfy._knb;
		else
			threads_z = MAX_THREADS_DIM;

		if(dom[dev].Gfy._inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfy._inb;
		else
			threads_x = MAX_THREADS_DIM;

		blocks_z = (int)ceil((real) dom[dev].Gfy._knb / (real) threads_z);
		blocks_x = (int)ceil((real) dom[dev].Gfy._inb / (real) threads_x);

		dim3 dimBlocks_y(threads_z, threads_x);
		dim3 numBlocks_y(blocks_z, blocks_x);
    
		// z-component
		if(dom[dev].Gfz._inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfz._inb;
		else
			threads_x = MAX_THREADS_DIM;

		if(dom[dev].Gfz._jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfz._jnb;
		else
			threads_y = MAX_THREADS_DIM;

		blocks_x = (int)ceil((real) dom[dev].Gfz._inb / (real) threads_x);
		blocks_y = (int)ceil((real) dom[dev].Gfz._jnb / (real) threads_y);

		dim3 dimBlocks_z(threads_x, threads_y);
		dim3 numBlocks_z(blocks_x, blocks_y);

		//need to do a linear interpolation to calculate the number density field on cell faces
		kernel_fz_coupling_numden_generate<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _numden[dev], _dom[dev]);

		//calculate numden[] on boundaries, this part will fail when multi-device domain decomposition is used.
		if(dom[dev].W == -1) {
			switch(numdenBC.nW) {
				case PERIODIC:
				BC_w_W_P<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].E == -1) {
			switch(numdenBC.nE) {
				case PERIODIC:
				BC_w_E_P<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].S == -1) {
			switch(numdenBC.nS) {
				case PERIODIC:
				BC_w_S_P<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].N == -1) {
			switch(numdenBC.nN) {
				case PERIODIC:
				BC_w_N_P<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].B == -1) {
			switch(numdenBC.nB) {
				case PERIODIC:
				BC_w_B_P<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _dom[dev]);
				break;
				case DIRICHLET:
				BC_w_B_D<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _dom[dev], numdenBC.nBD);
				break;
				case NEUMANN:
				BC_w_B_N<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].T == -1) {
			switch(numdenBC.nT) {
				case PERIODIC:
				BC_w_T_P<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _dom[dev]);
				break;
				case DIRICHLET:
				BC_w_T_D<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _dom[dev], numdenBC.nTD);
				break;
				case NEUMANN:
				BC_w_T_N<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev], _dom[dev]);
				break;
			}
		}
		
		// reset forcing arrays
		forcing_reset_x<<<numBlocks_x, dimBlocks_x>>>(_f_x[dev], _dom[dev]);
		forcing_reset_y<<<numBlocks_y, dimBlocks_y>>>(_f_y[dev], _dom[dev]);
		forcing_reset_z<<<numBlocks_z, dimBlocks_z>>>(_f_z[dev], _dom[dev]);
		
		// now add in the forcing
		real forcing_scale = -1.0/6.0*PI*(rho_f - bubble_density)*g.zm/rho_f;
		
		kernel_forcing_add_z_field_bubble<<<numBlocks_z, dimBlocks_z>>>(forcing_scale,
		                                                                _f_z_coupling_numden[dev],
		                                                                _bubdia[dev],
		                                                                _f_z[dev],
		                                                                _dom[dev]);
	}
}

extern "C"
void cuda_concentration_march(void)
{
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_y = 0;
		int threads_z = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		if(dom[dev].Gcc._jn < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc._jn;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc._kn < MAX_THREADS_DIM)
			threads_z = dom[dev].Gcc._kn;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gcc._jn / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gcc._kn / (real) threads_z);
		
		dim3 dimBlocks_c(threads_y, threads_z);
		dim3 numBlocks_c(blocks_y, blocks_z);
		
		// march the concentration equation
		kernel_march_concentration<<<numBlocks_c, dimBlocks_c>>>(dt,
                                                                 _dom[dev],
                                                                 _concen[dev],
                                                                 _nextconcen[dev],
                                                                 _u[dev],
                                                                 _v[dev],
                                                                 _w[dev],
                                                                 _mdot[dev],
                                                                 _numden[dev],
                                                                 concen_diff);
		
		// update concentration field
		kernel_inner_scalarfield_update_x<<<numBlocks_c, dimBlocks_c>>>(_dom[dev],
		                                                                 _concen[dev],
		                                                                 _nextconcen[dev]);
	}
}

extern "C"
void cuda_numberdensity_compute_totalnumden(void)
{
	// parallelize over CPU threads
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		
		// N is the number of inner cells
		int N = dom[dev].Gcc.s3;
		
		// create temporary storage for reduction algorithms
		real *_numden_tmp;
		checkCudaErrors(hipMalloc((void**) &_numden_tmp, sizeof(real) * N));
		gpumem += sizeof(real) * N;
		
		// set up kernel call
		if(dom[dev].Gcc.in < MAX_THREADS_DIM)
			threads_x = dom[dev].Gcc.in;
		else
			threads_x = MAX_THREADS_DIM;
			
		if(dom[dev].Gcc.jn < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc.jn;
		else
			threads_y = MAX_THREADS_DIM;
		
		blocks_x = (int)ceil((real) dom[dev].Gcc.in / (real) threads_x);
		blocks_y = (int)ceil((real) dom[dev].Gcc.jn / (real) threads_y);
		
		dim3 dimBlocks_z(threads_x, threads_y);
		dim3 numBlocks_z(blocks_x, blocks_y);
		
		// calculate summation
		kernel_numden_inner_copy<<<numBlocks_z, dimBlocks_z>>>(_dom[dev],
		                                                       _numden[dev],
		                                                       _numden_tmp);
		totalnumden = sum_entries(N, _numden_tmp);
		
		// clean up
		checkCudaErrors(hipFree(_numden_tmp));
		}
}

extern "C"
void cuda_bubblevolume_BC(void)
{
	// CPU threading for multi-GPU
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int threads_z = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		// check whether each subdomain boundary (E, W, N, S, T, B) is
		// an external boundary
		// *********************************************************************
		if(dom[dev].W == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks(threads_y, threads_z);
			dim3 numBlocks(blocks_y, blocks_z);
			
			// apply BC to bubble volume field for this face
			switch(bubvolBC.nW) {
				case PERIODIC:
					BC_p_W_P<<<numBlocks, dimBlocks>>>(_bubvol[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].E == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks(threads_y, threads_z);
			dim3 numBlocks(blocks_y, blocks_z);
			// apply BC to bubble volume for this face
			switch(bubvolBC.nE) {
				case PERIODIC:
					BC_p_E_P<<<numBlocks, dimBlocks>>>(_bubvol[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].S == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks(threads_z, threads_x);
			dim3 numBlocks(blocks_z, blocks_x);
			// apply BC to bubble volume for this face
			switch(bubvolBC.nS) {
				case PERIODIC:
					BC_p_S_P<<<numBlocks, dimBlocks>>>(_bubvol[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].N == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks(threads_z, threads_x);
			dim3 numBlocks(blocks_z, blocks_x);

			// apply BC to bubble volume for this face
			switch(bubvolBC.nN) {
					case PERIODIC:
					BC_p_N_P<<<numBlocks, dimBlocks>>>(_bubvol[dev], _dom[dev]);
				break;
			}
		}
		// *********************************************************************
		if(dom[dev].B == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks(threads_x, threads_y);
			dim3 numBlocks(blocks_x, blocks_y);
			// apply BC to bubble volume for this face
			switch(bubvolBC.nB) {
				case PERIODIC:
					BC_p_B_P<<<numBlocks, dimBlocks>>>(_bubvol[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_B_D<<<numBlocks, dimBlocks>>>(_bubvol[dev], _dom[dev], bubvolBC.nBD);
					break;
				case NEUMANN:
					BC_p_B_N<<<numBlocks, dimBlocks>>>(_bubvol[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].T == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks(threads_x, threads_y);
			dim3 numBlocks(blocks_x, blocks_y);
			// apply BC to bubble volume for this face
			switch(bubvolBC.nT) {
				case PERIODIC:
					BC_p_T_P<<<numBlocks, dimBlocks>>>(_bubvol[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_T_D<<<numBlocks, dimBlocks>>>(_bubvol[dev], _dom[dev], bubvolBC.nTD);
					break;
				case NEUMANN:
					BC_p_T_N<<<numBlocks, dimBlocks>>>(_bubvol[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		
		
	}
}

extern "C"
void cuda_compute_bubble_diameter(void)
{
	// parallelize over CPU threads
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int threads_z = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		// generate bubdia using bubvol(cell-centered field)
		if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc.jnb;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gcc.knb;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
		
		dim3 dimBlocks(threads_y, threads_z);
		dim3 numBlocks(blocks_y, blocks_z);
		
		kernel_compute_bubble_diameter<<<numBlocks, dimBlocks>>>(_dom[dev],
		                                                         _bubvol[dev],
		                                                         _bubdia[dev]);
		
		// interpolate bubdia to generate a z-face-centered field, which is needed in terminal velocity
		if(dom[dev].Gfz.inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfz.inb;
		else
			threads_x = MAX_THREADS_DIM;
		
		if(dom[dev].Gfz.jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfz.jnb;
		else
			threads_y = MAX_THREADS_DIM;
		
		blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
		blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
		
		dim3 dimBlocks_z(threads_x, threads_y);
		dim3 numBlocks_z(blocks_x, blocks_y);
		
		kernel_compute_bubble_diameterfz<<<numBlocks_z, dimBlocks_z>>>(_dom[dev],
		                                                               _bubdia[dev],
		                                                               _bubdiafz[dev]);
		
		if(dom[dev].W == -1) {
			switch(bubvolBC.nW) {
				case PERIODIC:
				BC_w_W_P<<<numBlocks_z, dimBlocks_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].E == -1) {
			switch(bubvolBC.nE) {
				case PERIODIC:
				BC_w_E_P<<<numBlocks_z, dimBlocks_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].S == -1) {
			switch(bubvolBC.nS) {
				case PERIODIC:
				BC_w_S_P<<<numBlocks_z, dimBlocks_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].N == -1) {
			switch(bubvolBC.nN) {
				case PERIODIC:
				BC_w_N_P<<<numBlocks_z, dimBlocks_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].B == -1) {
			switch(bubvolBC.nB) {
				case PERIODIC:
				BC_w_B_P<<<numBlocks_z, dimBlocks_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
				case DIRICHLET:
				BC_w_B_D<<<numBlocks_z, dimBlocks_z>>>(_bubdiafz[dev], _dom[dev], pow(6.0*bubvolBC.nBD/PI, 1.0/3.0));
				break;
				case NEUMANN:
				BC_w_B_N<<<numBlocks_z, dimBlocks_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].T == -1) {
			switch(bubvolBC.nT) {
				case PERIODIC:
				BC_w_T_P<<<numBlocks_z, dimBlocks_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
				case DIRICHLET:
				BC_w_T_D<<<numBlocks_z, dimBlocks_z>>>(_bubdiafz[dev], _dom[dev], pow(6.0*bubvolBC.nTD/PI, 1.0/3.0));
				break;
				case NEUMANN:
				BC_w_T_N<<<numBlocks_z, dimBlocks_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
	}
}

extern "C"
void cuda_compute_mdot(void)
{
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_y = 0;
		int threads_z = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		if(dom[dev].Gcc._jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc._jnb;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc._knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gcc._knb;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gcc._jnb / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gcc._knb / (real) threads_z);
		
		dim3 dimBlocks(threads_y, threads_z);
		dim3 numBlocks(blocks_y, blocks_z);
		
		kernel_compute_mdot<<<numBlocks, dimBlocks>>>(_dom[dev],
													  _concen[dev],
													  _bubdia[dev],
													  _mdot[dev],
													  _velmag[dev],
													  _u[dev],
													  _v[dev],
													  _w[dev],
													  concen_diss,
													  concen_diff,
													  nu);
	}
}

extern "C"
void cuda_bubblevolume_march(void)
{
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_y = 0;
		int threads_z = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		if(dom[dev].Gcc._jn < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc._jn;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc._kn < MAX_THREADS_DIM)
			threads_z = dom[dev].Gcc._kn;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gcc._jn / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gcc._kn / (real) threads_z);
		
		dim3 dimBlocks(threads_y, threads_z);
		dim3 numBlocks(blocks_y, blocks_z);
		
		// march the bubble volume equation
		kernel_march_bubblevolume<<<numBlocks, dimBlocks>>>(dt,
                                                            _dom[dev],
                                                            _bubvol[dev],
                                                            _nextbubvol[dev],
                                                            _u[dev],
                                                            _v[dev],
                                                            _w_p[dev],
                                                            _mdot[dev],
                                                            bubble_density);
		
		// update concentration field
		kernel_inner_scalarfield_update_x<<<numBlocks, dimBlocks>>>(_dom[dev],
		                                                            _bubvol[dev],
		                                                            _nextbubvol[dev]);
	}
}
