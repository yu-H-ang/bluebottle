#include "hip/hip_runtime.h"
#include "cuda_Eulerian.h"
#include "cuda_bluebottle.h"
#include "bluebottle.h"
#include "Eulerian.h"
#include "entrySearch.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C"
void cuda_Eulerian_push(void)
{
	int i, j, k;          // iterators
	int ii, jj, kk;       // helper iterators
	int C, CC;            // cell references
	
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		// set up host working arrays for subdomain copy from host to device
		real *nn = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		real *bubm = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		real *cc = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		//real *uu_p = (real*) malloc(dom[dev].Gfx.s3b * sizeof(real));
		//real *vv_p = (real*) malloc(dom[dev].Gfy.s3b * sizeof(real));
		//real *ww_p = (real*) malloc(dom[dev].Gfz.s3b * sizeof(real));
		
		// number density
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					nn[CC] = numden[C];
				}
			}
		}
		/*
		// u_p
		for(k = dom[dev].Gfx.ksb; k < dom[dev].Gfx.keb; k++) {
			for(j = dom[dev].Gfx.jsb; j < dom[dev].Gfx.jeb; j++) {
				for(i = dom[dev].Gfx.isb; i < dom[dev].Gfx.ieb; i++) {
					ii = i - dom[dev].Gfx.isb;
					jj = j - dom[dev].Gfx.jsb;
					kk = k - dom[dev].Gfx.ksb;
					C = i + j * Dom.Gfx.s1b + k * Dom.Gfx.s2b;
					CC = ii + jj * dom[dev].Gfx.s1b + kk * dom[dev].Gfx.s2b;
					uu_p[CC] = u_p[C];
				}
			}
		}
		// v_p
		for(k = dom[dev].Gfy.ksb; k < dom[dev].Gfy.keb; k++) {
			for(j = dom[dev].Gfy.jsb; j < dom[dev].Gfy.jeb; j++) {
				for(i = dom[dev].Gfy.isb; i < dom[dev].Gfy.ieb; i++) {
					ii = i - dom[dev].Gfy.isb;
					jj = j - dom[dev].Gfy.jsb;
					kk = k - dom[dev].Gfy.ksb;
					C = i + j * Dom.Gfy.s1b + k * Dom.Gfy.s2b;
					CC = ii + jj * dom[dev].Gfy.s1b + kk * dom[dev].Gfy.s2b;
					vv_p[CC] = v_p[C];
				}
			}
		}
		// w_p
		for(k = dom[dev].Gfz.ksb; k < dom[dev].Gfz.keb; k++) {
			for(j = dom[dev].Gfz.jsb; j < dom[dev].Gfz.jeb; j++) {
				for(i = dom[dev].Gfz.isb; i < dom[dev].Gfz.ieb; i++) {
					ii = i - dom[dev].Gfz.isb;
					jj = j - dom[dev].Gfz.jsb;
					kk = k - dom[dev].Gfz.ksb;
					C = i + j * Dom.Gfz.s1b + k * Dom.Gfz.s2b;
					CC = ii + jj * dom[dev].Gfz.s1b + kk * dom[dev].Gfz.s2b;
					ww_p[CC] = w_p[C];
				}
			}
		}
		*/
		// bubble mass
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					bubm[CC] = bubmas[C];
				}
			}
		}
		// concentration
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					cc[CC] = concen[C];
				}
			}
		}
		
		// copy from host to device
		checkCudaErrors(hipMemcpy(_numden[dev], nn, sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(_bubmas[dev], bubm, sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy(_concen[dev], cc, sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyHostToDevice));
		//checkCudaErrors(hipMemcpy(_u_p[dev], uu_p, sizeof(real) * dom[dev].Gfx.s3b, hipMemcpyHostToDevice));
		//checkCudaErrors(hipMemcpy(_v_p[dev], vv_p, sizeof(real) * dom[dev].Gfy.s3b, hipMemcpyHostToDevice));
		//checkCudaErrors(hipMemcpy(_w_p[dev], ww_p, sizeof(real) * dom[dev].Gfz.s3b, hipMemcpyHostToDevice));
		
		// free host subdomain working arrays
		free(nn);
		free(bubm);
		free(cc);
		//free(uu_p);
		//free(vv_p);
		//free(ww_p);
	}
}

void cuda_Eulerian_pull(void)
{
	// copy device data to host
	#pragma omp parallel num_threads(nsubdom)
	{
		int i, j, k;          // iterators
		int ii, jj, kk;       // helper iterators
		int C, CC;            // cell references
		
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));

		// host working arrays for subdomain copy from device to host
		real *nn = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		real *bubm = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		real *cc = (real*) malloc(dom[dev].Gcc.s3b * sizeof(real));
		//real *uu_p = (real*) malloc(dom[dev].Gfx.s3b * sizeof(real));
		//real *vv_p = (real*) malloc(dom[dev].Gfy.s3b * sizeof(real));
		//real *ww_p = (real*) malloc(dom[dev].Gfz.s3b * sizeof(real));
		
		// copy from device to host
		checkCudaErrors(hipMemcpy(nn, _numden[dev], sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(bubm, _bubmas[dev], sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyDeviceToHost));
		checkCudaErrors(hipMemcpy(cc, _concen[dev], sizeof(real) * dom[dev].Gcc.s3b, hipMemcpyDeviceToHost));
		//checkCudaErrors(hipMemcpy(uu_p, _u_p[dev], sizeof(real) * dom[dev].Gfx.s3b, hipMemcpyDeviceToHost));
		//checkCudaErrors(hipMemcpy(vv_p, _v_p[dev], sizeof(real) * dom[dev].Gfy.s3b, hipMemcpyDeviceToHost));
		//checkCudaErrors(hipMemcpy(ww_p, _w_p[dev], sizeof(real) * dom[dev].Gfz.s3b, hipMemcpyDeviceToHost));
		
		// fill in apropriate subdomain (copy back ghost cells)
		// numden
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					numden[C] = nn[CC];
				}
			}
		}
		/*
		// u
		for(k = dom[dev].Gfx.ksb; k < dom[dev].Gfx.keb; k++) {
			for(j = dom[dev].Gfx.jsb; j < dom[dev].Gfx.jeb; j++) {
				for(i = dom[dev].Gfx.isb; i < dom[dev].Gfx.ieb; i++) {
					ii = i - dom[dev].Gfx.isb;
					jj = j - dom[dev].Gfx.jsb;
					kk = k - dom[dev].Gfx.ksb;
					C = i + j * Dom.Gfx.s1b + k * Dom.Gfx.s2b;
					CC = ii + jj * dom[dev].Gfx.s1b + kk * dom[dev].Gfx.s2b;
					u_p[C] = uu_p[CC];
				}
			}
		}
		// v
		for(k = dom[dev].Gfy.ksb; k < dom[dev].Gfy.keb; k++) {
			for(j = dom[dev].Gfy.jsb; j < dom[dev].Gfy.jeb; j++) {
				for(i = dom[dev].Gfy.isb; i < dom[dev].Gfy.ieb; i++) {
					ii = i - dom[dev].Gfy.isb;
					jj = j - dom[dev].Gfy.jsb;
					kk = k - dom[dev].Gfy.ksb;
					C = i + j * Dom.Gfy.s1b + k * Dom.Gfy.s2b;
					CC = ii + jj * dom[dev].Gfy.s1b + kk * dom[dev].Gfy.s2b;
					v_p[C] = vv_p[CC];
				}
			}
		}
		// w
		for(k = dom[dev].Gfz.ksb; k < dom[dev].Gfz.keb; k++) {
			for(j = dom[dev].Gfz.jsb; j < dom[dev].Gfz.jeb; j++) {
				for(i = dom[dev].Gfz.isb; i < dom[dev].Gfz.ieb; i++) {
					ii = i - dom[dev].Gfz.isb;
					jj = j - dom[dev].Gfz.jsb;
					kk = k - dom[dev].Gfz.ksb;
					C = i + j * Dom.Gfz.s1b + k * Dom.Gfz.s2b;
					CC = ii + jj * dom[dev].Gfz.s1b + kk * dom[dev].Gfz.s2b;
					w_p[C] = ww_p[CC];
				}
			}
		}
		*/
		// bubmas
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					bubmas[C] = bubm[CC];
				}
			}
		}
		// concen
		for(k = dom[dev].Gcc.ksb; k < dom[dev].Gcc.keb; k++) {
			for(j = dom[dev].Gcc.jsb; j < dom[dev].Gcc.jeb; j++) {
				for(i = dom[dev].Gcc.isb; i < dom[dev].Gcc.ieb; i++) {
					ii = i - dom[dev].Gcc.isb;
					jj = j - dom[dev].Gcc.jsb;
					kk = k - dom[dev].Gcc.ksb;
					C = i + j * Dom.Gcc.s1b + k * Dom.Gcc.s2b;
					CC = ii + jj * dom[dev].Gcc.s1b + kk * dom[dev].Gcc.s2b;
					concen[C] = cc[CC];
				}
			}
		}
		
		// free host subdomain working arrays
		free(nn);
		free(bubm);
		free(cc);
		//free(uu_p);
		//free(vv_p);
		//free(ww_p);
	}
}

extern "C"
void cuda_Eulerian_free(void)
{
	// free device memory on device
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		checkCudaErrors(hipFree(_numden[dev]));
		checkCudaErrors(hipFree(_nextnumden[dev]));
		checkCudaErrors(hipFree(_w_b[dev]));
		checkCudaErrors(hipFree(_f_z_coupling_numden[dev]));
		
		checkCudaErrors(hipFree(_bubmas[dev]));
		checkCudaErrors(hipFree(_nextbubmas[dev]));
		checkCudaErrors(hipFree(_bubdia[dev]));
		checkCudaErrors(hipFree(_bubdiafz[dev]));
		
		checkCudaErrors(hipFree(_concen[dev]));
		checkCudaErrors(hipFree(_nextconcen[dev]));
		checkCudaErrors(hipFree(_velmag[dev]));
		checkCudaErrors(hipFree(_mdot[dev]));
	}
	
	// free device memory on host
	free(_numden);
	free(_nextnumden);
	free(_w_b);
	free(_f_z_coupling_numden);
	
	free(_bubmas);
	free(_nextbubmas);
	free(_bubdia);
	free(_bubdiafz);
	
	free(_concen);
	free(_nextconcen);
	free(_velmag);
	free(_mdot);
}

extern "C"
void cuda_numberdensity_BC(void)
{
	// CPU threading for multi-GPU
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int threads_z = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		// check whether each subdomain boundary (E, W, N, S, T, B) is
		// an external boundary
		// *********************************************************************
		if(dom[dev].W == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks_n(threads_y, threads_z);
			dim3 numBlocks_n(blocks_y, blocks_z);
			
			// apply BC to numberdensity field for this face
			switch(numdenBC.nW) {
				case PERIODIC:
					BC_p_W_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_W_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].E == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks_n(threads_y, threads_z);
			dim3 numBlocks_n(blocks_y, blocks_z);
			// apply BC to numberdensity field for this face
			switch(numdenBC.nE) {
				case PERIODIC:
					BC_p_E_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_E_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].S == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks_n(threads_z, threads_x);
			dim3 numBlocks_n(blocks_z, blocks_x);
			// apply BC to numberdensity field for this face
			switch(numdenBC.nS) {
				case PERIODIC:
					BC_p_S_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_S_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].N == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks_n(threads_z, threads_x);
			dim3 numBlocks_n(blocks_z, blocks_x);

			// apply BC to numberdensity field for this face
			switch(numdenBC.nN) {
					case PERIODIC:
					BC_p_N_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
				break;
					case NEUMANN:
					BC_p_N_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
				break;
			}
		}
		// *********************************************************************
		if(dom[dev].B == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks_n(threads_x, threads_y);
			dim3 numBlocks_n(blocks_x, blocks_y);
			// apply BC to numberdensity field for this face
			switch(numdenBC.nB) {
				case PERIODIC:
					BC_p_B_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_B_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_B_D<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev], numdenBC.nBD);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].T == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks_n(threads_x, threads_y);
			dim3 numBlocks_n(blocks_x, blocks_y);
			// apply BC to numberdensity field for this face
			switch(numdenBC.nT) {
				case PERIODIC:
					BC_p_T_P<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_T_N<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_T_D<<<numBlocks_n, dimBlocks_n>>>(_numden[dev], _dom[dev], numdenBC.nTD);
					break;
			}
		}
		// *********************************************************************
	}
}

extern "C"
void cuda_numberdensity_march(void)
{
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_y = 0;
		int threads_z = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		if(dom[dev].Gcc._jn < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc._jn;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc._kn < MAX_THREADS_DIM)
			threads_z = dom[dev].Gcc._kn;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gcc._jn / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gcc._kn / (real) threads_z);
		
		dim3 dimBlocks_n(threads_y, threads_z);
		dim3 numBlocks_n(blocks_y, blocks_z);
		
		kernel_march_numberdensity<<<numBlocks_n, dimBlocks_n>>>(dt,
		                                                         _dom[dev],
		                                                         _numden[dev],
		                                                         _nextnumden[dev],
		                                                         _u[dev],
		                                                         _v[dev],
		                                                         _w_b[dev]);
		
		kernel_inner_scalarfield_update_x<<<numBlocks_n, dimBlocks_n>>>(_dom[dev],
		                                                                _numden[dev],
		                                                                _nextnumden[dev]);
	}
}


extern "C"
void cuda_compute_particle_velz(void)
{
	// parallelize over CPU threads
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		
		// z-component
		if(dom[dev].Gfz._inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfz._inb;
		else
			threads_x = MAX_THREADS_DIM;
		if(dom[dev].Gfz._jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfz._jnb;
		else
			threads_y = MAX_THREADS_DIM;
		
		blocks_x = (int)ceil((real) dom[dev].Gfz._inb / (real) threads_x);
		blocks_y = (int)ceil((real) dom[dev].Gfz._jnb / (real) threads_y);
		dim3 dimBlocks_z(threads_x, threads_y);
		dim3 numBlocks_z(blocks_x, blocks_y);
		
		// since bubble diameter is a field, here combine all the values in terminal velocity except bubble diameter
		real cons = 1.0 / 18.0 * (rho_f - bubble_density) / mu * grav_acc;
		
		kernel_numberdensity_particle_velz<<<numBlocks_z, dimBlocks_z>>>(cons, _w_b[dev], _w[dev], _bubdiafz[dev], _dom[dev]);
	}
}

extern "C"
void cuda_Eulerian_malloc(void)
{
	// allocate device memory on host
	_numden = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_nextnumden = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	//_u_p = (real**) malloc(nsubdom * sizeof(real*));
	//cpumem += nsubdom * sizeof(real*);
	//_v_p = (real**) malloc(nsubdom * sizeof(real*));
	//cpumem += nsubdom * sizeof(real*);
	_w_b = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	//_f_x_coupling_numden = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	//_f_y_coupling_numden = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_f_z_coupling_numden = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	
	_bubmas = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_nextbubmas = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_bubdia = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_bubdiafz = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	
	_concen = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_nextconcen = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_velmag = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	_mdot = (real**) malloc(nsubdom * sizeof(real*));
	cpumem += nsubdom * sizeof(real*);
	
	// allocate device memory on device
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		// number density equation
		checkCudaErrors(hipMalloc((void**) &(_numden[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		// _numden_next is to store _numden in next time step, only exist on deivce.
		checkCudaErrors(hipMalloc((void**) &(_nextnumden[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		//checkCudaErrors(hipMalloc((void**) &(_u_p[dev]), sizeof(real) * dom[dev].Gfx.s3b));
		//gpumem += dom[dev].Gfx.s3b * sizeof(real);
		//checkCudaErrors(hipMalloc((void**) &(_v_p[dev]), sizeof(real) * dom[dev].Gfy.s3b));
		//gpumem += dom[dev].Gfy.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_w_b[dev]), sizeof(real) * dom[dev].Gfz.s3b));
		gpumem += dom[dev].Gfz.s3b * sizeof(real);
		//checkCudaErrors(hipMalloc((void**) &(_f_x_coupling_numden[dev]), sizeof(real) * dom[dev].Gfx.s3b));
		//gpumem += dom[dev].Gfx.s3b * sizeof(real);
		//checkCudaErrors(hipMalloc((void**) &(_f_y_coupling_numden[dev]), sizeof(real) * dom[dev].Gfy.s3b));
		//gpumem += dom[dev].Gfy.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_f_z_coupling_numden[dev]), sizeof(real) * dom[dev].Gfz.s3b));
		gpumem += dom[dev].Gfz.s3b * sizeof(real);
		
		// bubble mass
		checkCudaErrors(hipMalloc((void**) &(_bubmas[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_nextbubmas[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_bubdia[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_bubdiafz[dev]), sizeof(real) * dom[dev].Gfz.s3b));
		gpumem += dom[dev].Gfz.s3b * sizeof(real);
		
		// concentration
		checkCudaErrors(hipMalloc((void**) &(_concen[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		// _nextconcen is to store _concen in next time step, only exist on deivce.
		checkCudaErrors(hipMalloc((void**) &(_nextconcen[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		// _velmag is to store the velocity magnitude of the fluid flow,
		// which is used in the source term of mass transfer, only exist on device.
		checkCudaErrors(hipMalloc((void**) &(_velmag[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
		checkCudaErrors(hipMalloc((void**) &(_mdot[dev]), sizeof(real) * dom[dev].Gcc.s3b));
		gpumem += dom[dev].Gcc.s3b * sizeof(real);
	}
}

extern "C"
void cuda_concentration_BC(void)
{
	// CPU threading for multi-GPU
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int threads_z = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		// check whether each subdomain boundary (E, W, N, S, T, B) is
		// an external boundary
		// *********************************************************************
		if(dom[dev].W == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks_n(threads_y, threads_z);
			dim3 numBlocks_n(blocks_y, blocks_z);
			
			// apply BC to concentration field for this face
			switch(concenBC.nW) {
				case PERIODIC:
					BC_p_W_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_W_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].E == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks_n(threads_y, threads_z);
			dim3 numBlocks_n(blocks_y, blocks_z);
			// apply BC to concentration field for this face
			switch(concenBC.nE) {
				case PERIODIC:
					BC_p_E_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_E_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].S == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks_n(threads_z, threads_x);
			dim3 numBlocks_n(blocks_z, blocks_x);
			// apply BC to concentration field for this face
			switch(concenBC.nS) {
				case PERIODIC:
					BC_p_S_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_S_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].N == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks_n(threads_z, threads_x);
			dim3 numBlocks_n(blocks_z, blocks_x);

			// apply BC to concentration field for this face
			switch(concenBC.nN) {
					case PERIODIC:
					BC_p_N_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
				break;
					case NEUMANN:
					BC_p_N_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
				break;
			}
		}
		// *********************************************************************
		if(dom[dev].B == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks_n(threads_x, threads_y);
			dim3 numBlocks_n(blocks_x, blocks_y);
			// apply BC to concentration field for this face
			switch(concenBC.nB) {
				case PERIODIC:
					BC_p_B_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_B_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_B_D<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev], concenBC.nBD);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].T == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks_n(threads_x, threads_y);
			dim3 numBlocks_n(blocks_x, blocks_y);
			// apply BC to concentration field for this face
			switch(concenBC.nT) {
				case PERIODIC:
					BC_p_T_P<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case NEUMANN:
					BC_p_T_N<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_T_D<<<numBlocks_n, dimBlocks_n>>>(_concen[dev], _dom[dev], concenBC.nTD);
					break;
			}
		}
		// *********************************************************************
	}
}

extern "C"
void cuda_compute_coupling_forcing(void)
{
	// parallelize over CPU threads
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));

		int threads_x = 0;
		int threads_y = 0;
		int threads_z = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		int blocks_z = 0;

		// x-component
		if(dom[dev].Gfz._jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfz._jnb;
		else
			threads_y = MAX_THREADS_DIM;

		if(dom[dev].Gfz._knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gfz._knb;
		else
			threads_z = MAX_THREADS_DIM;

		blocks_y = (int)ceil((real) dom[dev].Gfz._jnb / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gfz._knb / (real) threads_z);

		dim3 dimBlocks_x(threads_y, threads_z);
		dim3 numBlocks_x(blocks_y, blocks_z);

		// y-component
		if(dom[dev].Gfz._knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gfz._knb;
		else
			threads_z = MAX_THREADS_DIM;

		if(dom[dev].Gfz._inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfz._inb;
		else
			threads_x = MAX_THREADS_DIM;

		blocks_z = (int)ceil((real) dom[dev].Gfz._knb / (real) threads_z);
		blocks_x = (int)ceil((real) dom[dev].Gfz._inb / (real) threads_x);

		dim3 dimBlocks_y(threads_z, threads_x);
		dim3 numBlocks_y(blocks_z, blocks_x);
    
		// z-component
		if(dom[dev].Gfz._inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfz._inb;
		else
			threads_x = MAX_THREADS_DIM;

		if(dom[dev].Gfz._jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfz._jnb;
		else
			threads_y = MAX_THREADS_DIM;

		blocks_x = (int)ceil((real) dom[dev].Gfz._inb / (real) threads_x);
		blocks_y = (int)ceil((real) dom[dev].Gfz._jnb / (real) threads_y);

		dim3 dimBlocks_z(threads_x, threads_y);
		dim3 numBlocks_z(blocks_x, blocks_y);
		
		// reset forcing arrays
		forcing_reset_x<<<numBlocks_x, dimBlocks_x>>>(_f_x[dev], _dom[dev]);
		forcing_reset_y<<<numBlocks_y, dimBlocks_y>>>(_f_y[dev], _dom[dev]);
		forcing_reset_z<<<numBlocks_z, dimBlocks_z>>>(_f_z[dev], _dom[dev]);
		
		// now add in the forcing
		real forcing_scale = 1.0/ 6.0 * PI * (rho_f - bubble_density) * grav_acc / rho_f;
		
		kernel_forcing_add_z_field_bubble<<<numBlocks_z, dimBlocks_z>>>(forcing_scale,
		                                                                _f_z_coupling_numden[dev],
		                                                                _bubdiafz[dev],
		                                                                _f_z[dev],
		                                                                _dom[dev]);
	}
}

extern "C"
void cuda_concentration_march(void)
{
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_y = 0;
		int threads_z = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		if(dom[dev].Gcc._jn < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc._jn;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc._kn < MAX_THREADS_DIM)
			threads_z = dom[dev].Gcc._kn;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gcc._jn / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gcc._kn / (real) threads_z);
		
		dim3 dimBlocks_c(threads_y, threads_z);
		dim3 numBlocks_c(blocks_y, blocks_z);
		
		// march the concentration equation
		kernel_march_concentration<<<numBlocks_c, dimBlocks_c>>>(dt,
                                                                 _dom[dev],
                                                                 _concen[dev],
                                                                 _nextconcen[dev],
                                                                 _u[dev],
                                                                 _v[dev],
                                                                 _w[dev],
                                                                 _mdot[dev],
                                                                 _numden[dev],
                                                                 concen_diff);
		
		// update concentration field
		kernel_inner_scalarfield_update_x<<<numBlocks_c, dimBlocks_c>>>(_dom[dev],
		                                                                 _concen[dev],
		                                                                 _nextconcen[dev]);
	}
}

extern "C"
void cuda_numberdensity_compute_totalnumden(void)
{
	// parallelize over CPU threads
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		
		// N is the number of inner cells
		int N = dom[dev].Gcc.s3;
		
		// create temporary storage for reduction algorithms
		real *_numden_tmp;
		checkCudaErrors(hipMalloc((void**) &_numden_tmp, sizeof(real) * N));
		gpumem += sizeof(real) * N;
		
		// set up kernel call
		if(dom[dev].Gcc.in < MAX_THREADS_DIM)
			threads_x = dom[dev].Gcc.in;
		else
			threads_x = MAX_THREADS_DIM;
			
		if(dom[dev].Gcc.jn < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc.jn;
		else
			threads_y = MAX_THREADS_DIM;
		
		blocks_x = (int)ceil((real) dom[dev].Gcc.in / (real) threads_x);
		blocks_y = (int)ceil((real) dom[dev].Gcc.jn / (real) threads_y);
		
		dim3 dimBlocks_z(threads_x, threads_y);
		dim3 numBlocks_z(blocks_x, blocks_y);
		
		// calculate summation
		kernel_numden_inner_copy<<<numBlocks_z, dimBlocks_z>>>(_dom[dev],
		                                                       _numden[dev],
		                                                       _numden_tmp);
		totalnumden = sum_entries(N, _numden_tmp);
		
		// clean up
		checkCudaErrors(hipFree(_numden_tmp));
		}
}

extern "C"
void cuda_bubblemass_BC(void)
{
	// CPU threading for multi-GPU
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int threads_z = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		// check whether each subdomain boundary (E, W, N, S, T, B) is
		// an external boundary
		// *********************************************************************
		if(dom[dev].W == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks(threads_y, threads_z);
			dim3 numBlocks(blocks_y, blocks_z);
			
			// apply BC to bubble volume field for this face
			switch(bubmasBC.nW) {
				case PERIODIC:
					BC_p_W_P<<<numBlocks, dimBlocks>>>(_bubmas[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].E == -1) {
			// set up kernel call
			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;
				
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;
			
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			
			dim3 dimBlocks(threads_y, threads_z);
			dim3 numBlocks(blocks_y, blocks_z);
			// apply BC to bubble volume for this face
			switch(bubmasBC.nE) {
				case PERIODIC:
					BC_p_E_P<<<numBlocks, dimBlocks>>>(_bubmas[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].S == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks(threads_z, threads_x);
			dim3 numBlocks(blocks_z, blocks_x);
			// apply BC to bubble volume for this face
			switch(bubmasBC.nS) {
				case PERIODIC:
					BC_p_S_P<<<numBlocks, dimBlocks>>>(_bubmas[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].N == -1) {
			// set up kernel call
			if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
				threads_z = dom[dev].Gcc.knb;
			else
				threads_z = MAX_THREADS_DIM;

			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);

			dim3 dimBlocks(threads_z, threads_x);
			dim3 numBlocks(blocks_z, blocks_x);

			// apply BC to bubble volume for this face
			switch(bubmasBC.nN) {
					case PERIODIC:
					BC_p_N_P<<<numBlocks, dimBlocks>>>(_bubmas[dev], _dom[dev]);
				break;
			}
		}
		// *********************************************************************
		if(dom[dev].B == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks(threads_x, threads_y);
			dim3 numBlocks(blocks_x, blocks_y);
			// apply BC to bubble volume for this face
			switch(bubmasBC.nB) {
				case PERIODIC:
					BC_p_B_P<<<numBlocks, dimBlocks>>>(_bubmas[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_B_D<<<numBlocks, dimBlocks>>>(_bubmas[dev], _dom[dev], bubmasBC.nBD);
					break;
				case NEUMANN:
					BC_p_B_N<<<numBlocks, dimBlocks>>>(_bubmas[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		if(dom[dev].T == -1) {
			// set up kernel call
			if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
				threads_x = dom[dev].Gcc.inb;
			else
				threads_x = MAX_THREADS_DIM;

			if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
				threads_y = dom[dev].Gcc.jnb;
			else
				threads_y = MAX_THREADS_DIM;

			blocks_x = (int)ceil((real) dom[dev].Gcc.inb / (real) threads_x);
			blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);

			dim3 dimBlocks(threads_x, threads_y);
			dim3 numBlocks(blocks_x, blocks_y);
			// apply BC to bubble volume for this face
			switch(bubmasBC.nT) {
				case PERIODIC:
					BC_p_T_P<<<numBlocks, dimBlocks>>>(_bubmas[dev], _dom[dev]);
					break;
				case DIRICHLET:
					BC_p_T_D<<<numBlocks, dimBlocks>>>(_bubmas[dev], _dom[dev], bubmasBC.nTD);
					break;
				case NEUMANN:
					BC_p_T_N<<<numBlocks, dimBlocks>>>(_bubmas[dev], _dom[dev]);
					break;
			}
		}
		// *********************************************************************
		
		
	}
}

extern "C"
void cuda_compute_bubble_diameter(void)
{
	// parallelize over CPU threads
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_x = 0;
		int threads_y = 0;
		int threads_z = 0;
		int blocks_x = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		real bottom_vol;
		
		//======================================================================
		// generate bubdia using bubmas(cell-centered field)
		
		if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc.jnb;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gcc.knb;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gcc.jnb / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gcc.knb / (real) threads_z);
		
		dim3 dimBlocks_Gcc_x(threads_y, threads_z);
		dim3 numBlocks_Gcc_x(blocks_y, blocks_z);
		
		kernel_compute_bubble_diameter<<<numBlocks_Gcc_x, dimBlocks_Gcc_x>>>(_dom[dev],
		                                                                     _bubmas[dev],
		                                                                     _numden[dev],
		                                                                     _bubdia[dev],
		                                                                     rho_f,
		                                                                     pressure_atm,
		                                                                     rho_atm,
		                                                                     grav_acc);
		
		//======================================================================
		// Do a linear interpolation to calculate the number density field on 
		// cell faces, which is needed when calculating face-centered bubble
		// diameter, and coupling force.
		
		// set up kernel call: Gfz-x-ghost
		if(dom[dev].Gfz._jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfz._jnb;
		else
			threads_y = MAX_THREADS_DIM;

		if(dom[dev].Gfz._knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gfz._knb;
		else
			threads_z = MAX_THREADS_DIM;

		blocks_y = (int)ceil((real) dom[dev].Gfz._jnb / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gfz._knb / (real) threads_z);

		dim3 dimBlocks_x(threads_y, threads_z);
		dim3 numBlocks_x(blocks_y, blocks_z);

		// set up kernel call: Gfz-y-ghost
		if(dom[dev].Gfz._knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gfz._knb;
		else
			threads_z = MAX_THREADS_DIM;

		if(dom[dev].Gfz._inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfz._inb;
		else
			threads_x = MAX_THREADS_DIM;

		blocks_z = (int)ceil((real) dom[dev].Gfz._knb / (real) threads_z);
		blocks_x = (int)ceil((real) dom[dev].Gfz._inb / (real) threads_x);

		dim3 dimBlocks_y(threads_z, threads_x);
		dim3 numBlocks_y(blocks_z, blocks_x);
		
		// set up kernel call: Gfz-z-ghost
		if(dom[dev].Gfz._inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfz._inb;
		else
			threads_x = MAX_THREADS_DIM;

		if(dom[dev].Gfz._jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfz._jnb;
		else
			threads_y = MAX_THREADS_DIM;

		blocks_x = (int)ceil((real) dom[dev].Gfz._inb / (real) threads_x);
		blocks_y = (int)ceil((real) dom[dev].Gfz._jnb / (real) threads_y);

		dim3 dimBlocks_z(threads_x, threads_y);
		dim3 numBlocks_z(blocks_x, blocks_y);
		
		kernel_fz_coupling_numden_generate<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev],
		                                                                 _numden[dev],
		                                                                 _dom[dev]);
		
		// Calculate numden on boundaries, this part will fail when multi-device
		// domain decomposition is used.
		if(dom[dev].W == -1) {
			switch(numdenBC.nW) {
				case PERIODIC:
				BC_w_W_P<<<numBlocks_x, dimBlocks_x>>>(_f_z_coupling_numden[dev],
				                                       _dom[dev]);
				break;
			}
		}
		if(dom[dev].E == -1) {
			switch(numdenBC.nE) {
				case PERIODIC:
				BC_w_E_P<<<numBlocks_x, dimBlocks_x>>>(_f_z_coupling_numden[dev],
				                                       _dom[dev]);
				break;
			}
		}
		if(dom[dev].S == -1) {
			switch(numdenBC.nS) {
				case PERIODIC:
				BC_w_S_P<<<numBlocks_y, dimBlocks_y>>>(_f_z_coupling_numden[dev],
				                                       _dom[dev]);
				break;
			}
		}
		if(dom[dev].N == -1) {
			switch(numdenBC.nN) {
				case PERIODIC:
				BC_w_N_P<<<numBlocks_y, dimBlocks_y>>>(_f_z_coupling_numden[dev],
				                                       _dom[dev]);
				break;
			}
		}
		if(dom[dev].B == -1) {
			switch(numdenBC.nB) {
				case PERIODIC:
				BC_w_B_P<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev],
				                                       _dom[dev]);
				break;
				case DIRICHLET:
				BC_w_B_D<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev],
				                                       _dom[dev], numdenBC.nBD);
				break;
				case NEUMANN:
				BC_w_B_N<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev],
				                                       _dom[dev]);
				break;
			}
		}
		if(dom[dev].T == -1) {
			switch(numdenBC.nT) {
				case PERIODIC:
					BC_w_T_P<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev],
				                                           _dom[dev]);
				break;
				case DIRICHLET:
					BC_w_T_D<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev],
				                                           _dom[dev],
				                                           numdenBC.nTD);
				break;
				case NEUMANN:
					BC_w_T_N<<<numBlocks_z, dimBlocks_z>>>(_f_z_coupling_numden[dev],
				                                           _dom[dev]);
				break;
			}
		}
		
		//======================================================================
		// Interpolate bubdia to generate a z-face-centered field, which is
		// needed in terminal velocity and coupling force.
		
		// set up kernel call: z
		if(dom[dev].Gfz.inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfz.inb;
		else
			threads_x = MAX_THREADS_DIM;
		
		if(dom[dev].Gfz.jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfz.jnb;
		else
			threads_y = MAX_THREADS_DIM;
		
		blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
		blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
		
		dim3 dimBlocks_Gfz_z(threads_x, threads_y);
		dim3 numBlocks_Gfz_z(blocks_x, blocks_y);
		
		kernel_compute_bubble_diameterfz<<<numBlocks_Gfz_z, dimBlocks_Gfz_z>>>(_dom[dev],
		                                                                       _bubmas[dev],
		                                                                       _f_z_coupling_numden[dev],
		                                                                       _bubdiafz[dev],
		                                                                       rho_f,
		                                                                       pressure_atm,
		                                                                       rho_atm,
		                                                                       grav_acc);
		
		// set up kernel call: x
		if(dom[dev].Gcc.jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gfz.jnb;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gfz.knb;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gfz.jnb / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gfz.knb / (real) threads_z);
		
		dim3 dimBlocks_Gfz_x(threads_y, threads_z);
		dim3 numBlocks_Gfz_x(blocks_y, blocks_z);
		
		// set up kernel call: y
		if(dom[dev].Gcc.knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gfz.knb;
		else
			threads_z = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc.inb < MAX_THREADS_DIM)
			threads_x = dom[dev].Gfz.inb;
		else
			threads_x = MAX_THREADS_DIM;
		
		blocks_z = (int)ceil((real) dom[dev].Gfz.knb / (real) threads_z);
		blocks_x = (int)ceil((real) dom[dev].Gfz.inb / (real) threads_x);
		
		dim3 dimBlocks_Gfz_y(threads_z, threads_x);
		dim3 numBlocks_Gfz_y(blocks_z, blocks_x);
		
		// impose boundary condition on z-face-centered number density field
		if(dom[dev].W == -1) {
			switch(bubmasBC.nW) {
				case PERIODIC:
					BC_w_W_P<<<numBlocks_Gfz_x, dimBlocks_Gfz_x>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].E == -1) {
			switch(bubmasBC.nE) {
				case PERIODIC:
					BC_w_E_P<<<numBlocks_Gfz_x, dimBlocks_Gfz_x>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].S == -1) {
			switch(bubmasBC.nS) {
				case PERIODIC:
					BC_w_S_P<<<numBlocks_Gfz_y, dimBlocks_Gfz_y>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].N == -1) {
			switch(bubmasBC.nN) {
				case PERIODIC:
					BC_w_N_P<<<numBlocks_Gfz_y, dimBlocks_Gfz_y>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].B == -1) {
			switch(bubmasBC.nB) {
				case PERIODIC:
					BC_w_B_P<<<numBlocks_Gfz_z, dimBlocks_Gfz_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
				case DIRICHLET:
					bottom_vol = bubmasBC.nBD / numdenBC.nBD / rho_atm / (1.0 + rho_f * grav_acc * dom[dev].zl / pressure_atm);
					BC_w_B_D<<<numBlocks_Gfz_z, dimBlocks_Gfz_z>>>(_bubdiafz[dev], _dom[dev], pow(6.0 * bottom_vol / PI, 1.0/3.0));
				break;
				case NEUMANN:
					BC_w_B_N<<<numBlocks_Gfz_z, dimBlocks_Gfz_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
			}
		}
		if(dom[dev].T == -1) {
			switch(bubmasBC.nT) {
				case PERIODIC:
					BC_w_T_P<<<numBlocks_Gfz_z, dimBlocks_Gfz_z>>>(_bubdiafz[dev], _dom[dev]);
				break;
				/*
				case DIRICHLET:
					BC_w_T_D<<<numBlocks_Gfz_z, dimBlocks_Gfz_z>>>(_bubdiafz[dev], _dom[dev], pow(6.0*(bubmasBC.nTD/rho_atm)/PI, 1.0/3.0));
				break;
				*/
				case NEUMANN:
					if(numdenBC.nT == NEUMANN) {
						BC_w_T_N<<<numBlocks_Gfz_z, dimBlocks_Gfz_z>>>(_bubdiafz[dev], _dom[dev]);
					} else {
						fprintf(stderr, "Boundary conditions of number density and bubble mass are not consistent at top.\n");
						exit(EXIT_FAILURE);
					}
				break;
			}
		}
	}
}

extern "C"
void cuda_compute_mdot(void)
{
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_y = 0;
		int threads_z = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		if(dom[dev].Gcc._jnb < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc._jnb;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc._knb < MAX_THREADS_DIM)
			threads_z = dom[dev].Gcc._knb;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gcc._jnb / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gcc._knb / (real) threads_z);
		
		dim3 dimBlocks(threads_y, threads_z);
		dim3 numBlocks(blocks_y, blocks_z);
		
		real cons = abs(1.0 / 18.0 * (rho_f - bubble_density) / mu * grav_acc);
		
		kernel_compute_mdot<<<numBlocks, dimBlocks>>>(_dom[dev],
                                                      _numden[dev],
                                                      _concen[dev],
                                                      _bubdia[dev],
                                                      _mdot[dev],
                                                      cons,
                                                      concen_diss,
                                                      concen_diff,
                                                      nu);
	}
}

extern "C"
void cuda_bubblemass_march(void)
{
	#pragma omp parallel num_threads(nsubdom)
	{
		int dev = omp_get_thread_num();
		checkCudaErrors(hipSetDevice(dev + dev_start));
		
		int threads_y = 0;
		int threads_z = 0;
		int blocks_y = 0;
		int blocks_z = 0;
		
		if(dom[dev].Gcc._jn < MAX_THREADS_DIM)
			threads_y = dom[dev].Gcc._jn;
		else
			threads_y = MAX_THREADS_DIM;
		
		if(dom[dev].Gcc._kn < MAX_THREADS_DIM)
			threads_z = dom[dev].Gcc._kn;
		else
			threads_z = MAX_THREADS_DIM;
		
		blocks_y = (int)ceil((real) dom[dev].Gcc._jn / (real) threads_y);
		blocks_z = (int)ceil((real) dom[dev].Gcc._kn / (real) threads_z);
		
		dim3 dimBlocks(threads_y, threads_z);
		dim3 numBlocks(blocks_y, blocks_z);
		
		// march the bubble mass equation
		kernel_march_bubblemass<<<numBlocks, dimBlocks>>>(dt,
                                                          _dom[dev],
                                                          _bubmas[dev],
                                                          _nextbubmas[dev],
                                                          _u[dev],
                                                          _v[dev],
                                                          _w_b[dev],
                                                          _numden[dev],
                                                          _mdot[dev]);
		
		// update bubmas field
		kernel_inner_scalarfield_update_x<<<numBlocks, dimBlocks>>>(_dom[dev],
		                                                            _bubmas[dev],
		                                                            _nextbubmas[dev]);
	}
}
